#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float* A,float* B, float* C, int N)
{
	int i = threadIdx.x;
	C[i]=A[i]+B[i];
}

int main()
{
	int N = 10;
	size_t size = N * sizeof(float);
	float* h_A = (float*)malloc(size);
	float* h_B = (float*)malloc(size);
	float* h_C = (float*)malloc(size);
	float* d_A;
	float* d_B;
	float* d_C;

	srand(time(NULL));
	
	for(i=0;i<N;i++)
	{
		h_A[i] = (float)(rand()%100)/10;
		h_B[i] = (float)(rand()%100)/10;
	}
	
	hipMalloc(&d_A,size);
	hipMalloc(&d_B,size);
	hipMalloc(&d_C,size);

	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

	VecAdd<<<1,N>>>(d_A,d_B,d_C,N);

	hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

	for(i=0;i<N;i++)
	{
		printf("%f",h_C[i]);
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}
