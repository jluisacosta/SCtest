#include "hip/hip_runtime.h"
/*
	PROGRAMA: Criba de Eratostenes (Obtencion de numeros primos) en CUDA.
	MATERIA: Supercomputo.
	AUTOR: Jose Luis Acosta Zamora (173025).
	FECHA: 25 Noviembre 2014.
*/

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

int* crea_arreglo(int n);
void imprime_arreglo(int n,int *vector);

__global__ void criba(int N,int tam_bloque,int desp,int **fragmento)
{
	int i,j,limite = (int)sqrt(N);

	for(i = 2 ;i <= limite; i++)
	{
		for(j = desp+2 ; j < (desp+2)+tam_bloque ; j++)
		{
			if(i!=j && j%i == 0)
			{
				*((*fragmento)+(j-desp-2))=-1;
			}
		}
	}
}

main(int argc, char *argv[])
{
	int N = atoi(*(argv+1))-1;
	int* h_lista;
	int* d_lista;
	int* h_primos;

	if(N>0)
	{
		printf("\nCRIBA DE ERATOSTENES\nObtencion de numeros primos entre 2 y %d\n",N+1);
		h_lista = crea_arreglo(N);
		h_primos = (int *)malloc(sizeof(int)*N); 

		criba<<<1,N>>>(d_A,d_B,d_C,N);
	
		printf("\n> RESULTADO : ");
		imprime_arreglo(N,primos);
		printf("\n");
	}
}

int* crea_arreglo(int n)
{
	int *a, i;

	a = (int *)malloc(sizeof(int)*n);
	for(i=0;i<n;*(a+i)=(i+2),i++);
	return a;
}

void imprime_arreglo(int n,int *arreglo)
{
	int i;

	for(i=0;i<n;i++)
	{
		if(*(arreglo+i)>=0)
		{
			printf("%d ",*(arreglo+i));
		}
	}
	printf("\n");
}


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float* A,float* B, float* C, int N)
{
	int i = threadIdx.x;
	C[i]=A[i]+B[i];
}

int main()
{
	int N = 10;
	size_t size = N * sizeof(float);
	float* h_A = (float*)malloc(size);
	float* h_B = (float*)malloc(size);
	float* h_C = (float*)malloc(size);
	float* d_A;
	float* d_B;
	float* d_C;

	srand(time(NULL));
	
	for(i=0;i<N;i++)
	{
		h_A[i] = (float)(rand()%100)/10;
		h_B[i] = (float)(rand()%100)/10;
	}
	
	hipMalloc(&d_A,size);
	hipMalloc(&d_B,size);
	hipMalloc(&d_C,size);

	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

	VecAdd<<<1,N>>>(d_A,d_B,d_C,N);

	hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

	for(i=0;i<N;i++)
	{
		printf("%f",h_C[i]);
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}
